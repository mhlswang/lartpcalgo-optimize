#include "hip/hip_runtime.h"

#include "hipfft/hipfft.h"
#include "utilities.h"

// cuda stuff
#define NREPS_PER_GPU 30
#define N_STREAMS 1

void make_plans(hipfftHandle* &plans, hipStream_t streams[], size_t wires_per_stream, size_t nticks);
void run_cufft(hipfftComplex* in, hipStream_t stream, size_t nticks, int nwires, int nreps);
void read_input_array_1D(hipfftReal** in_array, FILE* f, size_t nticks, size_t nwires, size_t nbatches);

//https://github.com/NVIDIA-developer-blog/code-samples
hipError_t checkCuda(hipError_t result);
hipfftResult checkCuFFT(hipfftResult r);

int main(int argc, char *argv[])
{


#ifdef USE_CALI
cali_id_t thread_attr = cali_create_attribute("thread_id", CALI_TYPE_INT, CALI_ATTR_ASVALUE | CALI_ATTR_SKIP_EVENTS);
#pragma omp parallel
{
cali_set_int(thread_attr, omp_get_thread_num());
}
#endif

  FILE* f;
  int nthr = 1;
  // omp_set_num_threads(nthr);

  int rep_to_check = 2;
  if (argc > 1) { rep_to_check = std::atoi(argv[1]); }
  //  f = fopen(argv[1], "r");
  // } else {
  //  f = fopen("noisefilt_100ev_50k.bin", "r");
  // }
  f = fopen("noisefilt_100ev_50k.bin", "r");
  if (f == NULL) {
    perror("Failed to open file: ");
    return 1;
  }

  size_t nticks = 4096;

  // hipEvent_t start_t, io_t1, fft_t, io_t2; 
  // hipEventCreate(&start_t);
  // hipEventCreate(&io_t1);
  // hipEventCreate(&fft_t);
  // hipEventCreate(&io_t2);

  // hipEventRecord(start_t);

  size_t nbatches = (int)std::trunc(NREPS/NREPS_PER_GPU) + 1;
  int leftover_reps = ( NREPS-NREPS_PER_GPU*(nbatches-1) );
  size_t nwires;

  fread(&nwires, sizeof(size_t), 1, f);

  std::cout << "found nwires    = " << nwires << std::endl;
  std::cout << "num reps        = " << NREPS << std::endl;
  std::cout << "num reps/gpu    = " << NREPS_PER_GPU << std::endl;
  std::cout << "extra reps      = " << leftover_reps << std::endl;
  std::cout << "num batches     = " << nbatches << std::endl;

  std::vector<std::vector<std::complex<float>> > expected_output;
  // expected_output.reserve(nwires);

  std::vector<std::vector<std::complex<float>> > computed_output;
  computed_output.reserve(nwires);
  for (int i = 0; i < nwires; ++i)
    computed_output[i].reserve(nticks);

  bool bad_mem = false;
  hipfftComplex **in;
  in = (hipfftComplex**)malloc(sizeof(hipfftComplex*) * nbatches);
  for(size_t r = 0; r < nbatches-1; r++) {
    in[r] = (hipfftComplex*)malloc(sizeof(hipfftComplex) * nwires * NREPS_PER_GPU * ((nticks/2+1)));
    bad_mem = bad_mem || (in[r] == NULL);
  }
  in[nbatches-1] = (hipfftComplex*)malloc(sizeof(hipfftComplex) * nwires * leftover_reps * ((nticks/2+1)) );
  bad_mem = bad_mem || (in[nbatches-1] == NULL);

  if (bad_mem) {
    std::cout << "ERROR: failed to malloc host data" << std::endl;
    exit(1);
  }
  
  read_input_array_1D((hipfftReal**)in, f, nticks, nwires, nbatches);
  read_output_vector(expected_output, f, nticks, nwires);
  fclose(f);
  // print_output_vector(expected_output, nticks);

  // hipEventRecord(io_t1);

  std::cout << "======================================================================================";
  std::cout << std::endl;
  std::cout << std::endl;
  std::cout << "Running cuFFT.....";   


  hipStream_t streams[N_STREAMS];
  for(size_t s = 0; s < N_STREAMS; s++) 
    checkCuda( hipStreamCreate(&streams[s]) );
 
  for(size_t r = 0; r < nbatches-1; r++){
    run_cufft(in[r], streams[0], nticks, nwires, NREPS_PER_GPU);
  }
  run_cufft(in[nbatches-1], streams[0], nticks, nwires, leftover_reps);

  // hipEventRecord(fft_t);

  std::cout << "DONE" << std::endl;
  std::cout << "======================================================================================";
  std::cout << std::endl;
  std::cout << std::endl;

  std::cout << "rep_to_check = " << rep_to_check << std::endl;


  // here we edit batch 0
  // "stream" 2
  // i.e. reps from reps_per_tream to 2* reps per stream
  // int reps_per_stream = NREPS_PER_GPU/N_STREAMS;
  // for (int i = 0; i < nwires * (reps_per_stream) * ((nticks/2+1)); ++i) {
  //   int offset = nwires * reps_per_stream * ((nticks/2+1)) * 2; 
  //   in[0][offset+i].x = 1000;
  //   in[0][offset+i].y = 1000;
  // }

  for (long iw=0; iw<nwires; ++iw) {
    for (long i = 0; i < nticks/2+1; ++i) {
      long idx = nwires * (nticks/2+1) * (rep_to_check%NREPS_PER_GPU) + iw*(nticks/2+1)+i; 

      // prints stuff fresh off the gpu
      // std::cout << "("   << hipCrealf(in[rep_to_check/NREPS_PER_GPU][idx]) 
      //           << " , " << hipCimagf(in[rep_to_check/NREPS_PER_GPU][idx]) << ")"
      //           << std::endl;

      computed_output[iw][i].real(hipCrealf(in[rep_to_check/NREPS_PER_GPU][idx])); 
      computed_output[iw][i].imag(hipCimagf(in[rep_to_check/NREPS_PER_GPU][idx]));
    }
    for (long j = 0; j < (nticks/2)+1; j++) {
      // computed_output[iw][(nticks/2)+j] = computed_output[iw][(nticks/2)-j];
      computed_output[iw][(nticks/2)+j] = std::conj(computed_output[iw][(nticks/2)-j]);
    }
  }

  hipFree(in);
  for(size_t r = 0; r < N_STREAMS; r++) 
    checkCuda( hipStreamDestroy(streams[r]) );
 
  #ifdef MAKE_PLOTS
  print_for_plots(PLOTS_FILE, expected_output, computed_output, nticks, nwires, true);
  #else
  print_err(expected_output, computed_output, nticks, nwires);
  #endif
  
  // hipEventRecord(io_t2);

  float t_tot = 0; //hipEventElapsedTime(&t_tot, start_t, io_t2);
  float t_io1 = 0; //hipEventElapsedTime(&t_io1, start_t, io_t1);
  float t_io2 = 0; //hipEventElapsedTime(&t_io2, fft_t,   io_t2);
  float t_fft = 0; //hipEventElapsedTime(&t_fft, io_t1,   fft_t);
  std::cout << "number thr = " << nthr << std::endl;
  std::cout << "total time = " << t_tot << "ms" << std::endl;
  std::cout << "io time    = " << t_io1 + t_io2 << "ms" << std::endl;
  std::cout << "fft time   = " << t_fft << "ms" << std::endl;
  std::cout << std::endl;


  hipDeviceReset();
  return 0;

}


void run_cufft(hipfftComplex* in, hipStream_t stream, size_t nticks, int nwires, int nreps) {

#ifdef USE_CALI
CALI_CXX_MARK_FUNCTION;
#endif

  hipfftHandle  plan;
  hipfftComplex* d_in;
  size_t size = sizeof(hipfftComplex) * nwires * nreps * ((nticks/2+1));
  checkCuda(hipHostRegister(in, size, hipHostRegisterPortable));
  bool bad_mem = false;
  checkCuda( hipMalloc(&d_in, size) );
  bad_mem = bad_mem || (d_in == NULL);

  if (bad_mem) {
    std::cout << "ERROR: failed to malloc device data" << std::endl;
    std::cout << "size = " << size << std::endl;
    std::cout << "size = " << nreps << std::endl;
    std::cout << "size = " << nwires << std::endl;
    exit(1);
  }

  // Make CUDA plans and streams

  int n = nticks;
  int istride = 1, ostride = 1;             // --- Distance between two successive input/output elements
  int idist = nticks+1, odist = nticks/2+1; // --- Distance between batches
  int inembed[] = { 0 };                    // --- Input size with pitch (ignored for 1D transforms)
  int onembed[] = { 0 };                    // --- Output size with pitch (ignored for 1D transforms)

  hipMemcpyAsync((void*)d_in, (void*)in, size, hipMemcpyHostToDevice, stream);

  checkCuFFT( hipfftPlanMany(&plan, 1, &n, 
              inembed, istride, idist, 
              onembed, ostride, odist, 
              HIPFFT_R2C, nwires*nreps) );
  checkCuFFT( hipfftSetStream(plan, stream) );

  checkCuFFT( hipfftExecR2C(plan, (hipfftReal*)d_in, d_in) );

  hipMemcpyAsync((void*)in, (void*)d_in, size, hipMemcpyDeviceToHost, stream);
  checkCuda( hipStreamSynchronize(stream) );

  // hipfftDestroy(plan);

  checkCuda(hipHostUnregister(in));
  hipFree(d_in);
 
}

void read_input_array_1D(hipfftReal** in_array, FILE* f, size_t nticks, size_t nwires, size_t nbatches) {

  if(in_array == NULL) std::cout << "in_array is NULL" << std::endl;

  for (size_t iw = 0; iw < nwires; ++iw) {
    for (size_t i = 0; i < nticks; ++i) {
      fread(&in_array[0][iw * (nticks+1) + i], sizeof(float), 1, f);
    }
  }
  
  for (size_t iw = nwires; iw < nwires * NREPS_PER_GPU; ++iw) {
    for (size_t i = 0; i < nticks; ++i) {
      in_array[0][iw * (nticks+1) + i] = in_array[0][(iw%nwires) * (nticks+1) + i];
    }
  }

  size_t r = 1;
  for (r = 1; r < nbatches-1; r++) {
    for (size_t iw = 0; iw < nwires * NREPS_PER_GPU; ++iw) {
      for (size_t i = 0; i < nticks; ++i) {
        in_array[r][iw * (nticks+1) + i] = in_array[0][iw * (nticks+1) + i];
      }
    }
  }
  r = nbatches-1;
  int leftover_wires = nwires * ( NREPS-NREPS_PER_GPU*(nbatches-1) ); 
  for (size_t iw = 0; iw < leftover_wires; ++iw) {
    for (size_t i = 0; i < nticks; ++i) {
      in_array[r][iw * (nticks+1) + i] = in_array[0][iw * (nticks+1) + i];
    }
  }

}



// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
// https://github.com/NVIDIA-developer-blog/code-samples
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

inline
hipfftResult checkCuFFT(hipfftResult r)
{
#if defined(DEBUG) || defined(_DEBUG)
  if(r != HIPFFT_SUCCESS) 
    std::cout << std::endl << "CUFFT ERROR:" << std::endl;
  if(r == HIPFFT_ALLOC_FAILED) 
    std::cout << "-- HIPFFT_ALLOC_FAILED" << std::endl;
  if(r == HIPFFT_INVALID_VALUE) 
    std::cout << "-- HIPFFT_INVALID_VALUE" << std::endl;
  if(r == HIPFFT_INTERNAL_ERROR) 
    std::cout << "-- HIPFFT_INTERNAL_ERROR" << std::endl;
  if(r == HIPFFT_SETUP_FAILED) 
    std::cout << "-- HIPFFT_SETUP_FAILED" << std::endl;
  if(r == HIPFFT_INVALID_PLAN) 
    std::cout << "-- HIPFFT_INVALID_PLAN" << std::endl;
  if(r == HIPFFT_EXEC_FAILED) 
    std::cout << "-- HIPFFT_EXEC_FAILED" << std::endl;
#endif
  return r;
}


