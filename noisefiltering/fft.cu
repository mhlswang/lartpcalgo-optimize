
#include "hipfft/hipfft.h"
#include "utilities.h"

// -I/usr/local/cuda/inc -L/usr/local/cuda/lib -lcufft

// cuda stuff for copy
#define TILE_DIM   32
#define BLOCK_ROWS 8

void run_cufft(float *in, hipfftComplex *out, 
              int nticks, int nwires, int nthr);

int main(int argc, char *argv[])
{


#ifdef USE_CALI
cali_id_t thread_attr = cali_create_attribute("thread_id", CALI_TYPE_INT, CALI_ATTR_ASVALUE | CALI_ATTR_SKIP_EVENTS);
#pragma omp parallel
{
cali_set_int(thread_attr, omp_get_thread_num());
}
#endif

  FILE* f;
  int nthr = 1;
  // omp_set_num_threads(nthr);

  if (argc > 1) {
    f = fopen(argv[1], "r");
  } else {
    f = fopen("noisefilt_100ev_50k.bin", "r");
  }
  
  assert(f);

  hipEvent_t start_t, io_t1, fft_t, io_t2; // change to cuda times?

  hipEventCreate(&start_t);
  hipEventCreate(&io_t1);
  hipEventCreate(&fft_t);
  hipEventCreate(&io_t2);

  hipEventRecord(start_t);

  size_t nwires;
  fread(&nwires, sizeof(size_t), 1, f);

  std::cout << "found nwires   =" << nwires << std::endl;
  std::cout << "number of reps =" << NREPS << std::endl;

  size_t nticks = 4096;

  float *in  = (float*) malloc(sizeof(float) * nticks * nwires);
  std::vector<std::vector<float> > input_vector;
  input_vector.reserve(nwires);

  std::vector<std::vector<std::complex<float>> > expected_output;
  expected_output.reserve(nwires);

  hipfftComplex *out  = (hipfftComplex*) malloc(sizeof(hipfftComplex) * nticks * nwires);
  std::vector<std::vector<std::complex<float>> > computed_output;
  computed_output.reserve(nwires);
  for (int i = 0; i < nwires; ++i)
    computed_output[i].resize(nticks);

  float *cu_in;
  hipfftComplex *cu_out;
  hipMalloc((void**)&cu_in, sizeof(float)*nticks*nwires);
  hipMalloc((void**)&cu_out, sizeof(hipfftComplex)*nticks*nwires);

  read_input_vector(input_vector, f, nticks, nwires);
  // print_input_vector(input_vector, nticks);
  for (int iw=0; iw<nwires; ++iw){
    for (int i = 0; i < nticks; ++i) in[iw*nticks+i] = input_vector[iw][i];
  }
  hipMemcpy(cu_in,in,sizeof(float)*nticks*nwires,hipMemcpyHostToDevice);
  

  std::cout << "======================================================================================";
  std::cout << std::endl;
  std::cout << std::endl;
  std::cout << "Running cuFFT.....";   

  hipEventRecord(io_t1);

  for (int i = 0; i < NREPS; i++) {

    run_cufft(cu_in, cu_out, nticks, nwires, nthr);

  }

  hipEventRecord(fft_t);

  std::cout << "DONE" << std::endl;
  std::cout << "======================================================================================";
  std::cout << std::endl;
  std::cout << std::endl;

  hipMemcpy(out,cu_out,sizeof(hipfftComplex)*nticks*nwires,hipMemcpyDeviceToHost);  
  for (int iw=0; iw<nwires; ++iw) {
    for (int i = 0; i < nticks/2+1; ++i) {
      computed_output[iw][i].real(hipCrealf(out[iw*nticks+i]));
      computed_output[iw][i].imag(hipCimagf(out[iw*nticks+i]));
    }
    for (int j = 0; j < (nticks/2)+1; j++) {
      computed_output[iw][(nticks/2)+j] = std::conj(computed_output[iw][(nticks/2)-j]);
    }
  }
  hipFree(cu_out);
  hipFree(cu_in);
  free(out);
  free(in);

  read_output_vector(expected_output, f, nticks, nwires);
  // print_output_vector(expected_output, nticks);

  fclose(f);
 
  #ifdef MAKE_PLOTS
  print_for_plots(PLOTS_FILE, expected_output, computed_output, nticks, nwires, true);
  #else
  print_err(expected_output, computed_output, nticks, nwires);
  #endif

  hipEventRecord(io_t2);

  float t_tot = 0; hipEventElapsedTime(&t_tot, start_t, io_t2);
  float t_io1 = 0; hipEventElapsedTime(&t_io1, start_t, io_t1);
  float t_io2 = 0; hipEventElapsedTime(&t_io2, fft_t,   io_t2);
  float t_fft = 0; hipEventElapsedTime(&t_fft, io_t1,   fft_t);
  std::cout << "number thr = " << nthr << std::endl;
  std::cout << "total time = " << t_tot << "ms" << std::endl;
  std::cout << "io time    = " << t_io1 + t_io2 << "ms" << std::endl;
  std::cout << "fft time   = " << t_fft << "ms" << std::endl;
  std::cout << std::endl;

  return 0;

}



void run_cufft(float *in, hipfftComplex *out, 
              int nticks, int nwires, int nthr) {

#ifdef USE_CALI
CALI_CXX_MARK_FUNCTION;
#endif


#define RANK 1


  hipfftHandle plan;
  // hipfftHandle iplan;

    int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
    int idist = nticks, odist = nticks; // --- Distance between batches
    int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
    int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
    
  // hipfftPlan1d(&plan, nticks, HIPFFT_R2C, nwires);
  hipfftPlanMany(&plan, RANK, &nticks, 
                inembed, istride, idist, 
                onembed, ostride, odist, 
                HIPFFT_R2C, nwires);
  // hipfftPlanMany(&iplan, RANK, &nticks, NULL, 0, 0, NULL, 0, 0, HIPFFT_C2R, nwires);


  // for (int iw=0; iw<nwires; ++iw) {

    // for (int i = 0; i < nticks; ++i) in[i] = input_vector[iw][i];

  hipfftExecR2C(plan, in, out); // R2C?
  hipDeviceSynchronize();

    // hipfftExecC2R(iplan, out, in); // C2R?
    // hipDeviceSynchronize();
  // }

  hipfftDestroy(plan);
  // hipfftDestroy(iplan);
  
}
