#include "hip/hip_runtime.h"

#include "hipfft/hipfft.h"
#include "utilities.h"

// cuda stuff
#define NREPS_PER_GPU 30
#define N_STREAMS 4

void make_plans(hipfftHandle* &plans, hipStream_t streams[], size_t wires_per_stream, size_t nticks);
void run_cufft(hipfftComplex* in, hipStream_t streams[], size_t nticks, int nwires, int nreps);
void read_input_array_1D(hipfftReal** in_array, FILE* f, size_t nticks, size_t nwires, size_t nbatches);

//https://github.com/NVIDIA-developer-blog/code-samples
hipError_t checkCuda(hipError_t result);
hipfftResult checkCuFFT(hipfftResult r);

int main(int argc, char *argv[])
{


#ifdef USE_CALI
cali_id_t thread_attr = cali_create_attribute("thread_id", CALI_TYPE_INT, CALI_ATTR_ASVALUE | CALI_ATTR_SKIP_EVENTS);
#pragma omp parallel
{
cali_set_int(thread_attr, omp_get_thread_num());
}
#endif

  FILE* f;
  int nthr = 1;
  // omp_set_num_threads(nthr);

  int rep_to_check = 2;
  if (argc > 1) { rep_to_check = std::atoi(argv[1]); }
  //  f = fopen(argv[1], "r");
  // } else {
  //  f = fopen("noisefilt_100ev_50k.bin", "r");
  // }
  f = fopen("noisefilt_100ev_50k.bin", "r");
  if (f == NULL) {
    perror("Failed to open file: ");
    return 1;
  }

  size_t nticks = 4096;

  // hipEvent_t start_t, io_t1, fft_t, io_t2; 
  // hipEventCreate(&start_t);
  // hipEventCreate(&io_t1);
  // hipEventCreate(&fft_t);
  // hipEventCreate(&io_t2);

  // hipEventRecord(start_t);

  size_t nbatches = (int)std::trunc(NREPS/NREPS_PER_GPU) + 1;
  int leftover_reps = ( NREPS-NREPS_PER_GPU*(nbatches-1) );
  size_t nwires;

  fread(&nwires, sizeof(size_t), 1, f);

  std::cout << "found nwires = " << nwires << std::endl;
  std::cout << "num reps     = " << NREPS << std::endl;
  std::cout << "num reps/gpu = " << NREPS_PER_GPU << std::endl;
  std::cout << "extra reps   = " << leftover_reps << std::endl;
  std::cout << "num batches  = " << nbatches << std::endl;

  std::vector<std::vector<std::complex<float>> > expected_output;
  // expected_output.reserve(nwires);

  std::vector<std::vector<std::complex<float>> > computed_output;
  computed_output.reserve(nwires);
  for (int i = 0; i < nwires; ++i)
    computed_output[i].reserve(nticks);

  bool bad_mem = false;
  hipfftComplex **in;
  in = (hipfftComplex**)malloc(sizeof(hipfftComplex*) * nbatches);
  for(size_t r = 0; r < nbatches-1; r++) {
    in[r] = (hipfftComplex*)malloc(sizeof(hipfftComplex) * nwires * NREPS_PER_GPU * ((nticks/2+1)));
    bad_mem = bad_mem || (in[r] == NULL);
  }
  in[nbatches-1] = (hipfftComplex*)malloc(sizeof(hipfftComplex) * nwires * leftover_reps * ((nticks/2+1)) );
  bad_mem = bad_mem || (in[nbatches-1] == NULL);

  if (bad_mem) {
    std::cout << "ERROR: failed to malloc host data" << std::endl;
    exit(1);
  }
  
  read_input_array_1D((hipfftReal**)in, f, nticks, nwires, nbatches);
  read_output_vector(expected_output, f, nticks, nwires);
  fclose(f);
  // print_output_vector(expected_output, nticks);

  // hipEventRecord(io_t1);

  std::cout << "======================================================================================";
  std::cout << std::endl;
  std::cout << std::endl;
  std::cout << "Running cuFFT.....";   


  hipStream_t streams[N_STREAMS];
  for(size_t s = 0; s < N_STREAMS; s++) 
    checkCuda( hipStreamCreate(&streams[s]) );
 
  for(size_t r = 0; r < nbatches-1; r++)
    run_cufft(in[r], streams, nticks, nwires, NREPS_PER_GPU);
  run_cufft(in[nbatches-1], streams, nticks, nwires, leftover_reps);

  checkCuda( hipDeviceSynchronize() ); 

  // hipEventRecord(fft_t);

  std::cout << "DONE" << std::endl;
  std::cout << "======================================================================================";
  std::cout << std::endl;
  std::cout << std::endl;

  std::cout << "rep_to_check = " << rep_to_check << std::endl;
  for (long iw=0; iw<nwires; ++iw) {
    for (long i = 0; i < nticks/2+1; ++i) {
      long idx = nwires * (nticks/2+1) * (rep_to_check%NREPS_PER_GPU) + iw*(nticks/2+1)+i; 
      computed_output[iw][i].real(hipCrealf(in[rep_to_check/NREPS_PER_GPU][idx])); 
      computed_output[iw][i].imag(hipCimagf(in[rep_to_check/NREPS_PER_GPU][idx]));
    }
    for (long j = 0; j < (nticks/2)+1; j++) {
      computed_output[iw][(nticks/2)+j] = std::conj(computed_output[iw][(nticks/2)-j]);
    }
  }

  hipFree(in);
  for(size_t r = 0; r < N_STREAMS; r++) 
    checkCuda( hipStreamDestroy(streams[r]) );
 
  #ifdef MAKE_PLOTS
  print_for_plots(PLOTS_FILE, expected_output, computed_output, nticks, nwires, true);
  #else
  print_err(expected_output, computed_output, nticks, nwires);
  #endif
  
  // hipEventRecord(io_t2);

  float t_tot = 0; //hipEventElapsedTime(&t_tot, start_t, io_t2);
  float t_io1 = 0; //hipEventElapsedTime(&t_io1, start_t, io_t1);
  float t_io2 = 0; //hipEventElapsedTime(&t_io2, fft_t,   io_t2);
  float t_fft = 0; //hipEventElapsedTime(&t_fft, io_t1,   fft_t);
  std::cout << "number thr = " << nthr << std::endl;
  std::cout << "total time = " << t_tot << "ms" << std::endl;
  std::cout << "io time    = " << t_io1 + t_io2 << "ms" << std::endl;
  std::cout << "fft time   = " << t_fft << "ms" << std::endl;
  std::cout << std::endl;


  hipDeviceReset();
  return 0;

}


void run_cufft(hipfftComplex* in, hipStream_t streams[], size_t nticks, int nwires, int nreps) {

#ifdef USE_CALI
CALI_CXX_MARK_FUNCTION;
#endif

  hipfftHandle  plans[N_STREAMS];

  int reps_per_stream = (int)std::trunc(nreps/N_STREAMS);
  int leftover_reps   = nreps%(N_STREAMS);
  int extra_reps      = reps_per_stream + leftover_reps;

  int size, offset;

  bool bad_mem = false;
  hipfftComplex** d_in = (hipfftComplex**)malloc((sizeof(hipfftComplex*) * N_STREAMS));
  // checkCuda( hipMalloc(&d_in, (sizeof(hipfftComplex*) * nbatches)) );
  for(size_t r = 0; r < N_STREAMS-1; r++) {
    checkCuda( hipMalloc(&d_in[r], sizeof(hipfftComplex) * nwires * reps_per_stream * ((nticks/2+1))) );
    bad_mem = bad_mem || (d_in[r] == NULL);
  }
  checkCuda( hipMalloc(&d_in[N_STREAMS-1], sizeof(hipfftComplex) * nwires * extra_reps * (nticks/2+1)) );
  bad_mem = bad_mem || (d_in[N_STREAMS-1] == NULL);

  if (bad_mem) {
    std::cout << "ERROR: failed to malloc device data" << std::endl;
    exit(1);
  }

  // Make CUDA plans and streams

  int n = nticks;
  int istride = 1, ostride = 1;             // --- Distance between two successive input/output elements
  int idist = nticks+1, odist = nticks/2+1; // --- Distance between batches
  int inembed[] = { 0 };                    // --- Input size with pitch (ignored for 1D transforms)
  int onembed[] = { 0 };                    // --- Output size with pitch (ignored for 1D transforms)

  for(size_t r = 0; r < N_STREAMS-1; r++){
    size   = sizeof(hipfftComplex) * nwires * reps_per_stream * ((nticks/2+1));
    offset = nwires * reps_per_stream * ((nticks/2+1)) * r;
    hipMemcpyAsync((void*)d_in[r], (void*)&in[offset], size, hipMemcpyHostToDevice, streams[r]);
  }
  size = sizeof(hipfftComplex) * nwires * extra_reps * (nticks/2+1);
  offset =  nwires * reps_per_stream * ((nticks/2+1)) * (N_STREAMS-1);
  hipMemcpyAsync((void*)d_in[N_STREAMS-1], (void*)&in[offset], size, hipMemcpyHostToDevice, streams[N_STREAMS-1]);
  
  for (int s = 0; s < N_STREAMS; s++) {

    int wires = nwires*reps_per_stream;
    if(s == N_STREAMS-1) wires = nwires*extra_reps;

    checkCuFFT( hipfftPlanMany(&plans[s], 1, &n, 
                inembed, istride, idist, 
                onembed, ostride, odist, 
                HIPFFT_R2C, wires) );
    checkCuFFT( hipfftSetStream(plans[s], streams[s]) );

    checkCuFFT( hipfftExecR2C(plans[s], (hipfftReal*)d_in[s], d_in[s]) );


  }
  for(size_t r = 0; r < N_STREAMS; r++)
    checkCuda( hipStreamSynchronize(streams[r]) );

  for(size_t r = 0; r < N_STREAMS-1; r++){
    size   = sizeof(hipfftComplex) * nwires * reps_per_stream * ((nticks/2+1));
    offset = nwires * reps_per_stream * ((nticks/2+1)) * r;
    hipMemcpyAsync((void*)&in[offset], (void*)d_in[r], size, hipMemcpyDeviceToHost, streams[r]);
  }
  size = sizeof(hipfftComplex) * nwires * extra_reps * (nticks/2+1);
  offset = nwires * reps_per_stream * ((nticks/2+1)) * (N_STREAMS-1);
  hipMemcpyAsync((void*)&in[offset], (void*)d_in[N_STREAMS-1], size, hipMemcpyDeviceToHost, streams[N_STREAMS-1]);

  for(size_t r = 0; r < N_STREAMS; r++)
    checkCuda( hipStreamSynchronize(streams[r]) );

  for(size_t r = 0; r < N_STREAMS; r++) {
    hipFree(d_in[r]);
    hipfftDestroy(plans[r]);
  }
  hipFree(d_in);
 
}

void read_input_array_1D(hipfftReal** in_array, FILE* f, size_t nticks, size_t nwires, size_t nbatches) {

  if(in_array == NULL) std::cout << "in_array is NULL" << std::endl;

  for (size_t iw = 0; iw < nwires; ++iw) {
    for (size_t i = 0; i < nticks; ++i) {
      fread(&in_array[0][iw * (nticks+1) + i], sizeof(float), 1, f);
    }
  }
  
  for (size_t iw = nwires; iw < nwires * NREPS_PER_GPU; ++iw) {
    for (size_t i = 0; i < nticks; ++i) {
      in_array[0][iw * (nticks+1) + i] = in_array[0][(iw%nwires) * (nticks+1) + i];
    }
  }

  size_t r = 1;
  for (r = 1; r < nbatches-1; r++) {
    for (size_t iw = 0; iw < nwires * NREPS_PER_GPU; ++iw) {
      for (size_t i = 0; i < nticks; ++i) {
        in_array[r][iw * (nticks+1) + i] = in_array[0][iw * (nticks+1) + i];
      }
    }
  }
  r = nbatches-1;
  int leftover_wires = nwires * ( NREPS-NREPS_PER_GPU*(nbatches-1) ); 
  for (size_t iw = 0; iw < leftover_wires; ++iw) {
    for (size_t i = 0; i < nticks; ++i) {
      in_array[r][iw * (nticks+1) + i] = in_array[0][iw * (nticks+1) + i];
    }
  }

}



// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
// https://github.com/NVIDIA-developer-blog/code-samples
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

inline
hipfftResult checkCuFFT(hipfftResult r)
{
#if defined(DEBUG) || defined(_DEBUG)
  if(r != HIPFFT_SUCCESS) 
    std::cout << std::endl << "CUFFT ERROR:" << std::endl;
  if(r == HIPFFT_ALLOC_FAILED) 
    std::cout << "-- HIPFFT_ALLOC_FAILED" << std::endl;
  if(r == HIPFFT_INVALID_VALUE) 
    std::cout << "-- HIPFFT_INVALID_VALUE" << std::endl;
  if(r == HIPFFT_INTERNAL_ERROR) 
    std::cout << "-- HIPFFT_INTERNAL_ERROR" << std::endl;
  if(r == HIPFFT_SETUP_FAILED) 
    std::cout << "-- HIPFFT_SETUP_FAILED" << std::endl;
  if(r == HIPFFT_INVALID_PLAN) 
    std::cout << "-- HIPFFT_INVALID_PLAN" << std::endl;
  if(r == HIPFFT_EXEC_FAILED) 
    std::cout << "-- HIPFFT_EXEC_FAILED" << std::endl;
#endif
  return r;
}


