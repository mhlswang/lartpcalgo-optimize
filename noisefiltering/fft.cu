
#include "hipfft/hipfft.h"
#include "utilities.h"

// cuda stuff for copy
#define TILE_DIM   32
#define BLOCK_ROWS 8

//will need to be tuned but needs to be < NREPS currently
#define N_STREAMS 2

void make_plans(hipfftHandle* &plans, hipStream_t streams[], size_t wires_per_stream, size_t nticks);
void run_cufft(hipfftComplex* in, size_t nticks, size_t nwires);
void read_input_array_1D(hipfftReal* in_array, FILE* f, size_t nticks, size_t nwires);

//https://github.com/NVIDIA-developer-blog/code-samples
hipError_t checkCuda(hipError_t result);
hipfftResult checkCuFFT(hipfftResult r);

int main(int argc, char *argv[])
{


#ifdef USE_CALI
cali_id_t thread_attr = cali_create_attribute("thread_id", CALI_TYPE_INT, CALI_ATTR_ASVALUE | CALI_ATTR_SKIP_EVENTS);
#pragma omp parallel
{
cali_set_int(thread_attr, omp_get_thread_num());
}
#endif

  FILE* f;
  int nthr = 1;
  // omp_set_num_threads(nthr);

  if (argc > 1) {
    f = fopen(argv[1], "r");
  } else {
    f = fopen("noisefilt_100ev_50k.bin", "r");
  }
  
  assert(f);

  size_t nticks = 4096;

  hipEvent_t start_t, io_t1, cp_t1, fft_t, io_t2, cp_t2; 
  hipEventCreate(&start_t);
  hipEventCreate(&io_t1);
  hipEventCreate(&cp_t1);
  hipEventCreate(&fft_t);
  hipEventCreate(&cp_t2);
  hipEventCreate(&io_t2);

  hipEventRecord(start_t);

  size_t nwires;
  fread(&nwires, sizeof(size_t), 1, f);

  std::cout << "found nwires     = " << nwires << std::endl;
  std::cout << "number of reps   = " << NREPS << std::endl;


  std::vector<std::vector<std::complex<float>> > expected_output;
  expected_output.reserve(nwires);

  std::vector<std::vector<std::complex<float>> > computed_output;
  computed_output.reserve(nwires);
  for (int i = 0; i < nwires; ++i)
    computed_output[i].resize(nticks);

  hipfftComplex* in;
  hipMallocManaged(&in, sizeof(hipfftComplex) * nwires * (nticks/2+1) * NREPS);
  read_input_array_1D((hipfftReal*)in, f, nticks, nwires);
  read_output_vector(expected_output, f, nticks, nwires);
  fclose(f);
  // print_output_vector(expected_output, nticks);

  hipEventRecord(io_t1);

  std::cout << "======================================================================================";
  std::cout << std::endl;
  std::cout << std::endl;
  std::cout << "Running cuFFT.....";   

  hipEventRecord(cp_t1);

  run_cufft(in, nticks, nwires);

  hipEventRecord(fft_t);

  std::cout << "DONE" << std::endl;
  std::cout << "======================================================================================";
  std::cout << std::endl;
  std::cout << std::endl;

  hipEventRecord(cp_t2);


  for (long iw=0; iw<nwires; ++iw) {
    for (long i = 0; i < nticks/2+1; ++i) {
      int idx = nwires * (nticks/2+1) * 5 + iw*(nticks/2+1)+i; // 5 is an arbitrary REP to grab
      computed_output[iw][i].real(hipCrealf(in[idx])); 
      computed_output[iw][i].imag(hipCimagf(in[idx]));
    }
    for (long j = 0; j < (nticks/2)+1; j++) {
      computed_output[iw][(nticks/2)+j] = std::conj(computed_output[iw][(nticks/2)-j]);
    }
  }

  hipFree(in);
 
  #ifdef MAKE_PLOTS
  print_for_plots(PLOTS_FILE, expected_output, computed_output, nticks, nwires, true);
  #else
  print_err(expected_output, computed_output, nticks, nwires);
  #endif
  
  hipEventRecord(io_t2);

  float t_tot = 0; hipEventElapsedTime(&t_tot, start_t, io_t2);
  float t_io1 = 0; hipEventElapsedTime(&t_io1, start_t, io_t1);
  float t_io2 = 0; hipEventElapsedTime(&t_io2, cp_t2,   io_t2);
  float t_cp1 = 0; hipEventElapsedTime(&t_cp1, io_t1,   cp_t1);
  float t_cp2 = 0; hipEventElapsedTime(&t_cp2, fft_t,   cp_t2);
  float t_fft = 0; hipEventElapsedTime(&t_fft, cp_t1,   fft_t);
  std::cout << "number thr = " << nthr << std::endl;
  std::cout << "total time = " << t_tot << "ms" << std::endl;
  std::cout << "io time    = " << t_io1 + t_io2 << "ms" << std::endl;
  std::cout << "copy time  = " << t_cp1 + t_cp2 << "ms" << std::endl;
  std::cout << "fft time   = " << t_fft << "ms" << std::endl;
  std::cout << std::endl;


  hipDeviceReset();
  return 0;

}


void run_cufft(hipfftComplex* in,  
              size_t nticks, size_t nwires) {

#ifdef USE_CALI
CALI_CXX_MARK_FUNCTION;
#endif

  // Make CUDA plans and streams
  hipfftHandle plan;
  

  int n = nticks;
  int istride = 1, ostride = 1;             // --- Distance between two successive input/output elements
  int idist = nticks+1, odist = nticks/2+1; // --- Distance between batches
  int inembed[] = { 0 };                    // --- Input size with pitch (ignored for 1D transforms)
  int onembed[] = { 0 };                    // --- Output size with pitch (ignored for 1D transforms)
  int batches = nwires*NREPS;

  
  checkCuFFT( hipfftPlanMany(&plan, 1, &n, 
              inembed, istride, idist, 
              onembed, ostride, odist, 
              HIPFFT_R2C, batches) );
  
  checkCuFFT( hipfftExecR2C(plan, (hipfftReal*)in, in) );
  hipDeviceSynchronize();  
 
}

void read_input_array_1D(hipfftReal* in_array, FILE* f, size_t nticks, size_t nwires) {

  if(in_array == NULL) std::cout << "in_array is NULL" << std::endl;

  for (size_t iw = 0; iw < nwires; ++iw) {
    for (size_t i = 0; i < nticks; ++i) {
      fread(&in_array[iw * (nticks+1) + i], sizeof(float), 1, f);
    }
  }

  for (size_t iw = nwires; iw < nwires * NREPS; ++iw) {
    for (size_t i = 0; i < nticks; ++i) {
      in_array[iw * (nticks+1) + i] = in_array[iw%nwires * (nticks+1) + i];
    }
  }

}



// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
// https://github.com/NVIDIA-developer-blog/code-samples
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

inline
hipfftResult checkCuFFT(hipfftResult r)
{
#if defined(DEBUG) || defined(_DEBUG)
  if(r != HIPFFT_SUCCESS) 
    std::cout << std::endl << "CUFFT ERROR:" << std::endl;
  if(r == HIPFFT_ALLOC_FAILED) 
    std::cout << "-- HIPFFT_ALLOC_FAILED" << std::endl;
  if(r == HIPFFT_INVALID_VALUE) 
    std::cout << "-- HIPFFT_INVALID_VALUE" << std::endl;
  if(r == HIPFFT_INTERNAL_ERROR) 
    std::cout << "-- HIPFFT_INTERNAL_ERROR" << std::endl;
  if(r == HIPFFT_SETUP_FAILED) 
    std::cout << "-- HIPFFT_SETUP_FAILED" << std::endl;
  if(r == HIPFFT_INVALID_PLAN) 
    std::cout << "-- HIPFFT_INVALID_PLAN" << std::endl;
  if(r == HIPFFT_EXEC_FAILED) 
    std::cout << "-- HIPFFT_EXEC_FAILED" << std::endl;
#endif
  return r;
}


